
#include <hip/hip_runtime.h>
#ifndef THC_GENERIC_FILE
#define THC_GENERIC_FILE "THC/generic/THCTensorMathPairwise.cu"
#else

#include <ATen/NamedTensorUtils.h>

static int THCTensor_(equalImpl)(THCState *state, THCTensor *self_, THCTensor *src_)
{
  THCAssertSameGPU(THCTensor_(checkGPU)(state, 2, self_, src_));
  if (!THCTensor_(isSameSizeAs(state, self_, src_))) {
    return 0;
  }

  // This is not as efficient as TH, but the basic idea: create a buffer that stores
  // 1 if the two tensors are equal at a position, otherwise 0. If the minimum value
  // in this buffer is 1, the two tensors are equal, otherwise they are not

  // Both tensors are empty
  if(THTensor_(nElement)(self_) == 0) return true;

  THCudaByteTensor *buf = THCudaByteTensor_newWithSize(state, self_->sizes(), {});

  if (!THC_pointwiseApply3<uint8_t, scalar_t, scalar_t>(state, buf, self_, src_, TensorEQOp<scalar_t, unsigned char>())) {
    THArgCheck(false, 2, CUTORCH_DIM_WARNING);
  }

  unsigned char min = THCudaByteTensor_minall(state, buf);

  THCudaByteTensor_free(state, buf);

  return min != 0;
}

int THCTensor_(equal)(THCState *state, THCTensor *self_, THCTensor *src_) {
  if (!at::namedinference::are_names_equal(self_, src_)) {
    return 0;
  }
  at::NoNamesGuard guard;
  return THCTensor_(equalImpl)(state, self_, src_);
}

#if !defined(THC_REAL_IS_BOOL)

void THCTensor_(mul)(THCState *state, THCTensor *self_, THCTensor *src_, scalar_t value)
{
  THCAssertSameGPU(THCTensor_(checkGPU)(state, 2, self_, src_));
  if (self_ == src_) {
    if (!THC_pointwiseApply1<scalar_t>(state, self_, TensorMulConstantOp<scalar_t>(value))) {
      THArgCheck(false, 2, CUTORCH_DIM_WARNING);
    }
  } else {
    THCTensor_(resizeAs)(state, self_, src_);

    if (!THC_pointwiseApply2<scalar_t, scalar_t>(state, self_, src_, TensorMulConstantOp<scalar_t>(value))) {
      THArgCheck(false, 2, CUTORCH_DIM_WARNING);
    }
  }

  THCudaCheck(cudaGetLastError());
}

void THCTensor_(div)(THCState* state, THCTensor *self_, THCTensor *src_, scalar_t value)
{
  THCAssertSameGPU(THCTensor_(checkGPU)(state, 2, self_, src_));
  THArgCheck(value != ScalarConvert<int, scalar_t>::to(0), 3, "divide by zero");

  if (self_ == src_) {
    if (!THC_pointwiseApply1<scalar_t>(state, self_, TensorDivConstantOp<scalar_t>(value))) {
      THArgCheck(false, 2, CUTORCH_DIM_WARNING);
    }
  } else {
    THCTensor_(resizeAs)(state, self_, src_);

    if (!THC_pointwiseApply2<scalar_t, scalar_t>(state, self_, src_, TensorDivConstantOp<scalar_t>(value))) {
      THArgCheck(false, 2, CUTORCH_DIM_WARNING);
    }
  }

  THCudaCheck(cudaGetLastError());
}

void THCTensor_(rshift)(THCState* state, THCTensor *self_, THCTensor *src_, scalar_t value)
{
#if defined(THC_REAL_IS_FLOAT) || defined(THC_REAL_IS_DOUBLE)
  THCTensor_(mul)(state, self_, src_, pow(2, -value));
#elif defined(THC_REAL_IS_HALF)
  return THError("rshift not supported for torch.CudaHalfTensor");
#else
  if (self_ == src_) {
    if (!THC_pointwiseApply1<scalar_t>(state, self_, TensorRShiftConstantOp<scalar_t>(value))) {
      THArgCheck(false, 2, CUTORCH_DIM_WARNING);
    }
  } else {
    THCTensor_(resizeAs)(state, self_, src_);

    if (!THC_pointwiseApply2<scalar_t, scalar_t>(state, self_, src_, TensorRShiftConstantOp<scalar_t>(value))) {
      THArgCheck(false, 2, CUTORCH_DIM_WARNING);
    }
  }

  THCudaCheck(cudaGetLastError());
#endif
}

void THCTensor_(fmod)(THCState *state, THCTensor *self_, THCTensor *src_, scalar_t value)
{
  THCAssertSameGPU(THCTensor_(checkGPU)(state, 2, self_, src_));
  if (self_ == src_) {
    if (!THC_pointwiseApply1<scalar_t>(state, self_, TensorFmodOp<scalar_t>(value))) {
      THArgCheck(false, 2, CUTORCH_DIM_WARNING);
    }
  } else {
    THCTensor_(resizeAs)(state, self_, src_);

    if (!THC_pointwiseApply2<scalar_t, scalar_t>(state, self_, src_, TensorFmodOp<scalar_t>(value))) {
      THArgCheck(false, 2, CUTORCH_DIM_WARNING);
    }
  }

  THCudaCheck(cudaGetLastError());
}

void THCTensor_(remainder)(THCState *state, THCTensor *self_, THCTensor *src_, scalar_t value)
{
  THCAssertSameGPU(THCTensor_(checkGPU)(state, 2, self_, src_));
  if (self_ == src_) {
    if (!THC_pointwiseApply1<scalar_t>(state, self_, TensorRemainderOp<scalar_t>(value))) {
      THArgCheck(false, 2, CUTORCH_DIM_WARNING);
    }
  } else {
    THCTensor_(resizeAs)(state, self_, src_);

    if (!THC_pointwiseApply2<scalar_t, scalar_t>(state, self_, src_, TensorRemainderOp<scalar_t>(value))) {
      THArgCheck(false, 2, CUTORCH_DIM_WARNING);
    }
  }

  THCudaCheck(cudaGetLastError());
}

void THCTensor_(triu)(THCState *state, THCTensor *self_, THCTensor *src_, int64_t k)
{
  THCAssertSameGPU(THCTensor_(checkGPU)(state, 2, self_, src_));
  THArgCheck(!src_->is_empty() && src_->dim() == 2, 1, "expected a matrix");

  if (self_ != src_)
    THCTensor_(resizeAs)(state, self_, src_);

  int64_t stride0 = self_->stride(0);
  int64_t stride1 = self_->stride(1);
  scalar_t *start = THCTensor_(data)(state, self_);

  TensorTriOp<scalar_t, 1> op(start, stride0, stride1, k);

  if (self_ == src_) {
    if (!THC_pointwiseApply1<scalar_t>(state, src_, op)) {
      THArgCheck(false, 2, CUTORCH_DIM_WARNING);
    }
  } else {

    if (!THC_pointwiseApply2<scalar_t, scalar_t>(state, self_, src_, op)) {
      THArgCheck(false, 2, CUTORCH_DIM_WARNING);
    }
  }

  THCudaCheck(cudaGetLastError());
}

#endif

#endif
